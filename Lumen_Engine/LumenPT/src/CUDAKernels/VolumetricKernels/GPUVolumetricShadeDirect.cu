#include "hip/hip_runtime.h"
#include "GPUVolumetricShadingKernels.cuh"
#include <>
#include <sutil/vec_math.h>

using namespace WaveFront;

GPU_ONLY void VolumetricShadeDirect(
    const uint3 a_ResolutionAndDepth,
    const WaveFront::VolumetricData* a_VolumetricDataBuffer,
    WaveFront::AtomicBuffer<WaveFront::ShadowRayData>* const a_ShadowRays,
    const WaveFront::TriangleLight* const a_Lights,
    const unsigned int a_NumLights,
    const CDF* const a_CDF)
{

    return;

}