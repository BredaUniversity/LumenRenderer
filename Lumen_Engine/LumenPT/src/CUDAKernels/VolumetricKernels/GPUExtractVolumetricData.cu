#include "GPUVolumetricShadingKernels.cuh"
#include <>
#include <sutil/vec_math.h>

using namespace WaveFront;

CPU_ON_GPU void ExtractVolumetricDataGpu(
    unsigned a_NumIntersections,
    WaveFront::AtomicBuffer<WaveFront::IntersectionRayData>* a_Rays,
    WaveFront::AtomicBuffer<WaveFront::VolumetricIntersectionData>* a_IntersectionData,
    WaveFront::VolumetricData* a_OutPut,
    SceneDataTableAccessor* a_SceneDataTable)
{

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < a_NumIntersections; i += stride)
    {

        const VolumetricIntersectionData* currIntersection = a_IntersectionData->GetData(i);
        const IntersectionRayData* currRay = a_Rays->GetData(currIntersection->m_RayArrayIndex);
        unsigned int pixelIndex = currIntersection->m_PixelIndex;


        //TODO: for each intersection fill a VolumetricData struct and place in the right pixel index.
        //The struct with information will be used in the shading functions so should contain all the necessary data for this.
        //eg. incoming ray direction, entryIntersectionT, exitIntersectionT, position, etc.

        auto& output = a_OutPut[pixelIndex];
        output.m_PixelIndex = pixelIndex;
        output.m_PositionEntry = currRay->m_Origin + currRay->m_Direction * currIntersection->m_EntryT;
        output.m_PositionExit = currRay->m_Origin + currRay->m_Direction * currIntersection->m_ExitT;

    }

}