#include "hip/hip_runtime.h"
#include "ReSTIRKernels.cuh"

#include <>

#include "../../vendor/Include/Cuda/cuda/helpers.h"

__host__ void ResetReservoirs(int a_NumReservoirs, Reservoir* a_ReservoirPointer)
{
    //Call in parallel.
    const int blockSize = 256;
    const int numBlocks = (a_NumReservoirs + blockSize - 1) / blockSize;
    ResetReservoirInternal<<<numBlocks, blockSize>>>(a_NumReservoirs, a_ReservoirPointer);
}

__global__ void ResetReservoirInternal(int a_NumReservoirs, Reservoir* a_ReservoirPointer)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < a_NumReservoirs; i += stride)
    {
        a_ReservoirPointer[i].Reset();
    }
}
