#include "hip/hip_runtime.h"
#include "ReSTIRKernels.cuh"

#include "../Shaders/CppCommon/RenderingUtility.h"
#include "../Shaders/CppCommon/WaveFrontDataStructs.h"
#include <hip/hip_runtime_api.h>
#include <cuda/device_atomic_functions.h>

#define CUDA_BLOCK_SIZE 256

__host__ void ResetReservoirs(int a_NumReservoirs, Reservoir* a_ReservoirPointer)
{
    //Call in parallel.
    const int blockSize = CUDA_BLOCK_SIZE;
    const int numBlocks = (a_NumReservoirs + blockSize - 1) / blockSize;
    ResetReservoirInternal<<<numBlocks, blockSize>>>(a_NumReservoirs, a_ReservoirPointer);

    //TODO: Wait after every task may not be needed.Check if it is required between kernel calls.
    hipDeviceSynchronize();
}

__global__ void ResetReservoirInternal(int a_NumReservoirs, Reservoir* a_ReservoirPointer)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < a_NumReservoirs; i += stride)
    {
        a_ReservoirPointer[i].Reset();
    }
}

__host__ void FillCDF(CDF* a_Cdf, TriangleLight* a_Lights, unsigned a_LightCount)
{
    //TODO: This is not efficient single threaded.
    //TODO: Use this: https://developer.nvidia.com/gpugems/gpugems3/part-vi-gpu-computing/chapter-39-parallel-prefix-sum-scan-cuda

    //First reset the CDF on the GPU.
    ResetCDF<<<1,1>>>(a_Cdf);

    //Run from one thread because it's not thread safe to append the sum of each element.
    FillCDFInternal <<<1, 1>>> (a_Cdf, a_Lights, a_LightCount);
    hipDeviceSynchronize();
}

__global__ void ResetCDF(CDF* a_Cdf)
{
    a_Cdf->Reset();
}

__global__ void FillCDFInternal(CDF* a_Cdf, TriangleLight* a_Lights, unsigned a_LightCount)
{
    for (int i = 0; i < a_LightCount; ++i)
    {
        //Weight is the average illumination for now. Could take camera into account.
        const float3 radiance = a_Lights[i].radiance;
        a_Cdf->Insert((radiance.x + radiance.y + radiance.z) / 3.f);
    }
}

__host__ void FillLightBags(unsigned a_NumLightBags, CDF* a_Cdf, LightBagEntry* a_LightBagPtr, TriangleLight* a_Lights, const std::uint32_t a_Seed)
{
    const int blockSize = CUDA_BLOCK_SIZE;
    const int numBlocks = (a_NumLightBags + blockSize - 1) / blockSize;
    FillLightBagsInternal <<<numBlocks, blockSize >>>(a_NumLightBags, a_Cdf, a_LightBagPtr, a_Lights, a_Seed);
    hipDeviceSynchronize();
}

__global__ void FillLightBagsInternal(unsigned a_NumLightBags, CDF* a_Cdf, LightBagEntry* a_LightBagPtr, TriangleLight* a_Lights, const std::uint32_t a_Seed)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < a_NumLightBags; i += stride)
    {
        //Generate a random float between 0 and 1.
        auto seed = a_Seed + i;
        float random = RandomFloat(seed);

        //Store the pdf and light in the light bag.
        unsigned lIndex;
        float pdf;
        a_Cdf->Get(random, lIndex, pdf);
        a_LightBagPtr[i] = {a_Lights[lIndex], pdf};
    }
}

__host__ void PickPrimarySamples(const WaveFront::RayData* const a_RayData, const WaveFront::IntersectionData* const a_IntersectionData, const LightBagEntry* const a_LightBags, Reservoir* a_Reservoirs, const ReSTIRSettings& a_Settings, PixelData* a_PixelData, const std::uint32_t a_Seed)
{
    //TODO ensure that each pixel grid operates within a single block, and that the L1 cache is not overwritten for each value. Optimize for cache hits.
    //TODO correctly assign a light bag per grid through some random generation.

    const auto numReservoirs = (a_Settings.width * a_Settings.height * a_Settings.numReservoirsPerPixel);
    const int blockSize = CUDA_BLOCK_SIZE;
    const int numBlocks = (numReservoirs + blockSize - 1) / blockSize;
    PickPrimarySamplesInternal<<<numBlocks, blockSize>>>(a_RayData, a_IntersectionData, a_LightBags, a_Reservoirs, a_Settings, a_PixelData, a_Seed);
    hipDeviceSynchronize();
}

__global__ void PickPrimarySamplesInternal(const WaveFront::RayData* const a_RayData, const WaveFront::IntersectionData* const a_IntersectionData, const LightBagEntry* const a_LightBags, Reservoir* a_Reservoirs, const ReSTIRSettings& a_Settings, PixelData* a_PixelData, const std::uint32_t a_Seed)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    const auto numPixels = a_Settings.width * a_Settings.height;

    //Seed for this thread index.
    auto lightBagSeed = a_Seed + blockIdx.x;    //Seed is the same for each block so that they all get the same light bag.
    float random = RandomFloat(lightBagSeed);

    //Generate between 0 and 1, then round and pick a light bag index based on the total light bag amount.
    int lightBagIndex = static_cast<int>(round(static_cast<float>(a_Settings.numLightBags - 1) * random));

    auto* pickedLightBag = &a_LightBags[lightBagIndex * a_Settings.numLightsPerBag];

    //Loop over the pixels
    for (int i = index; i < numPixels; i += stride)
    {
        //Get the intersection data for this pixel.
        auto* intersectionData = &a_IntersectionData[i];

        //Extract the pixel data from the right buffers, and store them for this pixel index.
        PixelData* pixel = &a_PixelData[i];

        //If no intersection exists at this pixel, do nothing.
        if(intersectionData->m_IntersectionT <= 0.f)
        {
            //Set pixel depth to 0 to be able to identify unused pixels easily.
            pixel->depth = -1;
            continue;
        }

        //The ray that resulted in this intersection.
        auto* ray = &a_RayData[intersectionData->m_RayArrayIndex];

        //Extract the pixel features from all the different buffers. Store them in the pixel.
        pixel->worldPosition = ray->m_Origin + ray->m_Direction * intersectionData->m_IntersectionT;
        pixel->directionIncoming = ray->m_Direction;

        const unsigned int vertexIndex = 3 * intersectionData->m_PrimitiveIndex;
        const DevicePrimitive* primitive = intersectionData->m_Primitive;
        const unsigned int vertexIndexA = primitive->m_IndexBuffer[vertexIndex + 0];
        const unsigned int vertexIndexB = primitive->m_IndexBuffer[vertexIndex + 1];
        const unsigned int vertexIndexC = primitive->m_IndexBuffer[vertexIndex + 2];

        const Vertex* A = &primitive->m_VertexBuffer[vertexIndexA];
        const Vertex* B = &primitive->m_VertexBuffer[vertexIndexB];
        const Vertex* C = &primitive->m_VertexBuffer[vertexIndexC];

        const float U = intersectionData->m_UVs.x;
        const float V = intersectionData->m_UVs.y;
        const float W = 1.f - (U + V);

        const float2 texCoords = A->m_UVCoord * W + B->m_UVCoord * U + C->m_UVCoord * V;
        const float4 texColor = tex2D<float4>(primitive->m_Material->m_DiffuseTexture, texCoords.x, texCoords.y);
        const float4 triangleColor = primitive->m_Material->m_DiffuseColor;

        pixel->worldNormal = normalize(A->m_Normal + B->m_Normal + C->m_Normal);
        pixel->diffuse = float3{texColor.x * triangleColor.x, texColor.y * triangleColor.y, texColor.z * triangleColor.z};
        pixel->roughness = 1.f; //intersectionData->m_Primitive->m_Material->m_Roughness; //TODO
        pixel->metallic = 0.f;  //intersectionData->m_Primitive->m_Material->m_Metallic; //TODO


        pixel->depth = intersectionData->m_IntersectionT;

        //For every pixel, update each reservoir.
        for (int reservoirIndex = 0; reservoirIndex < a_Settings.numReservoirsPerPixel; ++reservoirIndex)
        {
            auto* reservoir = &a_Reservoirs[RESERVOIR_INDEX(i, reservoirIndex, a_Settings.numReservoirsPerPixel)];
            reservoir->Reset();

            //Only sample for intersected pixels.
            if (intersectionData->m_IntersectionT <= 0.f)
            {
                continue;
            }

            //Generate the amount of samples specified per reservoir.
            for (int sample = 0; sample < a_Settings.numPrimarySamples; ++sample)
            {
                //Random number using the pixel id.
                auto seed = a_Seed + i;
                float r = RandomFloat(seed);

                const int pickedLightIndex = static_cast<int>(round(static_cast<float>(a_Settings.numLightsPerBag - 1) * r));
                const LightBagEntry pickedEntry = pickedLightBag[pickedLightIndex];
                const TriangleLight light = pickedEntry.light;
                const float initialPdf = pickedEntry.pdf;

                //Generate random UV coordinates. Between 0 and 1.
                const float u = RandomFloat(seed);  //Seed is altered after each shift, which makes it work with the same uint.
                const float v = RandomFloat(seed);

                //Generate a sample with solid angle PDF for this specific pixel.
                LightSample lightSample;
                {
                    //Fill the light with the right settings.
                    lightSample.radiance = light.radiance;
                    lightSample.normal = light.normal;
                    lightSample.area = light.area;

                    //TODO generate random point according to UV coordinates. This is taking the center for now.
                    lightSample.position = (light.p0 + light.p1 + light.p2) / 3.f;

                    //Calculate the PDF for this pixel and light.
                    Resample(&lightSample, pixel, &lightSample);
                }

                //The final PDF for the light in this reservoir is the solid angle divided by the original PDF of the light being chosen based on radiance.
                const auto pdf = lightSample.solidAnglePdf / initialPdf;
                reservoir->Update(lightSample, pdf, a_Seed);
            }

            reservoir->UpdateWeight();
        }
    }
}

__host__ int GenerateReSTIRShadowRays(MemoryBuffer* a_AtomicCounter, Reservoir* a_Reservoirs, RestirShadowRay* a_ShadowRays, PixelData* a_PixelData)
{
    //Counter that is atomically incremented. Copy it to the GPU.
    int atomic = 0;
    a_AtomicCounter->Write(atomic);
    auto devicePtr = a_AtomicCounter->GetDevicePtr<int>();
    const auto numPixels = ReSTIRSettings::width * ReSTIRSettings::height;

    //Call in parallel.
    const int blockSize = CUDA_BLOCK_SIZE;
    const int numBlocks = (numPixels + blockSize - 1) / blockSize;
    GenerateShadowRay<<<numBlocks, blockSize>>> (devicePtr, a_Reservoirs, a_ShadowRays, a_PixelData);
    hipDeviceSynchronize();

    //Copy value back to the CPU.
    a_AtomicCounter->Read(&atomic, sizeof(int), 0);

    //Return the amount of rays that have been generated.
    return atomic;
}

__global__ void GenerateShadowRay(int* a_AtomicCounter, Reservoir* a_Reservoirs, RestirShadowRay* a_ShadowRays, PixelData* a_PixelData)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    const auto numPixels = ReSTIRSettings::width * ReSTIRSettings::height;

    for (int pixel = index; pixel < numPixels; pixel += stride)
    {
        PixelData* pixelData = &a_PixelData[pixel];        

        //Only run for valid intersections.
        if(pixelData->depth > 0.f)
        {
            float3 pixelPosition = pixelData->worldPosition;

            //Run for every reservoir for the pixel.
            for(int depth = 0; depth < ReSTIRSettings::numReservoirsPerPixel; ++depth)
            {
                //If the reservoir has a weight, add a shadow ray.
                Reservoir* reservoir = &a_Reservoirs[RESERVOIR_INDEX(pixel, depth, ReSTIRSettings::numReservoirsPerPixel)];
                if(reservoir->weight > 0.f)
                {

                    int shadowIndex = atomicAdd(a_AtomicCounter, 1);

                    float3 pixelToLight = (reservoir->sample.position - pixelPosition);
                    float l = length(pixelToLight);
                    pixelToLight /= l;

                    RestirShadowRay ray;
                    ray.index = pixel;
                    ray.direction = pixelToLight;
                    ray.origin = pixelPosition;
                    ray.distance = l - 0.005f; //Make length a little bit shorter to prevent self-shadowing.

                    a_ShadowRays[shadowIndex] = ray;
                }
            }            
        }
    }
}

__host__ void SpatialNeighbourSampling(Reservoir* a_Reservoirs, Reservoir* a_SwapBuffer, PixelData* a_PixelData, const std::uint32_t a_Seed)
{    
    const auto numPixels = (ReSTIRSettings::width * ReSTIRSettings::height);
    const int blockSize = CUDA_BLOCK_SIZE;
    const int numBlocks = (numPixels + blockSize - 1) / blockSize;
    SpatialNeighbourSamplingInternal<<<numBlocks, blockSize >>>(a_Reservoirs, a_SwapBuffer, a_PixelData, a_Seed);
    hipDeviceSynchronize();
}

//TODO access settings struct statically instead of passing an instance.

__global__ void SpatialNeighbourSamplingInternal(Reservoir* a_Reservoirs, Reservoir* a_SwapBuffer,
    PixelData* a_PixelData, const std::uint32_t a_Seed)
{
    Reservoir* fromBuffer = a_Reservoirs;
    Reservoir* toBuffer = a_SwapBuffer;

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    const auto numPixels = (ReSTIRSettings::width * ReSTIRSettings::height);

    //Storage for reservoirs and pixels to be combined.
    PixelData* toCombinePixelData[ReSTIRSettings::numSpatialSamples + 1];
    Reservoir* toCombineReservoirs[ReSTIRSettings::numSpatialSamples + 1];

    //Loop over the pixels.
    for (int i = index; i < numPixels; i += stride)
    {
        //The seed unique to this pixel.
        auto seed = a_Seed + i;

        toCombinePixelData[0] = &a_PixelData[i];

        //Only run when there's an intersection for this pixel.
        if (toCombinePixelData[0]->depth > 0.f)
        {
            //TODO maybe store this information inside the pixel. Could calculate it once at the start of the frame.
            const int y = i / ReSTIRSettings::width;
            const int x = i - (y * ReSTIRSettings::width);

            for (int iteration = 0; iteration < ReSTIRSettings::numSpatialIterations; ++iteration)
            {
                for (int depth = 0; depth < ReSTIRSettings::numReservoirsPerPixel; ++depth)
                {
                    toCombineReservoirs[0] = &a_Reservoirs[RESERVOIR_INDEX(i, depth, ReSTIRSettings::numReservoirsPerPixel)];

                    int count = 1;

                    //TODO move this loop up maybe? Use same neighbour for each depth since values are different anyways. I don't think that changes the
                    //TODO average of the calculation.
                    for (int neighbour = 1; neighbour <= ReSTIRSettings::numSpatialSamples; ++neighbour)
                    {
                        //TODO This generates a square rn. Make it within a circle.
                        const int neighbourY = round((RandomFloat(seed) * 2.f - 1.f) * static_cast<float>(ReSTIRSettings::spatialSampleRadius)) + y;
                        const int neighbourX = round((RandomFloat(seed) * 2.f - 1.f) * static_cast<float>(ReSTIRSettings::spatialSampleRadius)) + x;
                        const int neighbourIndex = PIXEL_INDEX(neighbourX, neighbourY, ReSTIRSettings::numReservoirsPerPixel);

                        //Only run if within image bounds.
                        if(neighbourX >= 0 && neighbourX <= ReSTIRSettings::width && neighbourY >= 0 && neighbourY <= ReSTIRSettings::height)
                        {
                            Reservoir* pickedReservoir = &a_Reservoirs[RESERVOIR_INDEX(neighbourIndex, depth, ReSTIRSettings::numReservoirsPerPixel)];
                            PixelData* pickedPixel = &a_PixelData[neighbourIndex];

                            //Only run for valid depths.
                            if(pickedPixel->depth > 0)
                            {
                                //Gotta stay positive.
                                assert(pickedReservoir->weight >= 0.f);

                                //Discard samples that are too different.
                                float depth1 = pickedPixel->depth;
                                float depth2 = toCombinePixelData[0]->depth;
                                float depthDifPct = fabs(depth1 - depth2) / ((depth1 + depth2) / 2.f);

                                const float angleDif = dot(pickedPixel->worldNormal, toCombinePixelData[0]->worldNormal);	//Between 0 and 1 (0 to 90 degrees). 
                                static constexpr float MAX_ANGLE_COS = 0.72222222223f;	//Dot product is cos of the angle. If higher than this value, it's within 25 degrees.

                                if (depthDifPct < 0.10f && angleDif > MAX_ANGLE_COS)
                                {
                                    toCombineReservoirs[count] = pickedReservoir;
                                    toCombinePixelData[count] = pickedPixel;
                                    ++count;
                                }
                            }
                        }
                    }

                    //If valid reservoirs to combine were found, combine them.
                    if (count > 1)
                    {
                        if(ReSTIRSettings::enableBiased)
                        {
                            CombineBiased(i, count, toCombineReservoirs, toCombinePixelData, seed);
                        }
                        else
                        {
                            CombineUnbiased(i, count, toCombineReservoirs, toCombinePixelData, seed);
                        }
                    }
                }

                //Swap the pointers for in and output.
                Reservoir* temp = fromBuffer;
                fromBuffer = toBuffer;
                toBuffer = temp;
            }
        }
    }
}


__host__ void TemporalNeighbourSampling(
    Reservoir* a_CurrentReservoirs,
    Reservoir* a_PreviousReservoirs,
    PixelData* a_CurrentPixelData,
    PixelData* a_PreviousPixelData,
    const std::uint32_t a_Seed
)
{
    const int numPixels = (ReSTIRSettings::width * ReSTIRSettings::height);
    const int blockSize = CUDA_BLOCK_SIZE;
    const int numBlocks = (numPixels + blockSize - 1) / blockSize;

    //TODO pass the motion vector information in here.

    CombineTemporalSamplesInternal << <numBlocks, blockSize >> > (a_CurrentReservoirs, a_PreviousReservoirs,
                                                                  a_CurrentPixelData, a_PreviousPixelData, a_Seed);
    hipDeviceSynchronize();
}


__global__ void CombineTemporalSamplesInternal(
    Reservoir* a_CurrentReservoirs,
    Reservoir* a_PreviousReservoirs,
    PixelData* a_CurrentPixelData,
    PixelData* a_PreviousPixelData,
    const std::uint32_t a_Seed
)
{
    const int numPixels = (ReSTIRSettings::width * ReSTIRSettings::height);
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    Reservoir* toCombine[2];
    PixelData* pixelPointers[2];

    for (int i = index; i < numPixels; i += stride)
    {
        //TODO instead look up the motion vector and use that to find the right pixel. This assumes a static scene rn.
        const int temporalIndex = i;

        pixelPointers[0] = &a_PreviousPixelData[temporalIndex];
        pixelPointers[1] = &a_CurrentPixelData[i];

        //Ensure that the depth of both samples is valid, and then combine them at each depth.
        if (pixelPointers[0]->depth > 0.f && pixelPointers[1]->depth > 0.f)
        {
            //For every reservoir at the current pixel.
            for (int depth = 0; depth < ReSTIRSettings::numReservoirsPerPixel; ++depth)
            {
                toCombine[0] = &a_PreviousReservoirs[RESERVOIR_INDEX(temporalIndex, depth, ReSTIRSettings::numReservoirsPerPixel)];
                toCombine[1] = &a_CurrentReservoirs[RESERVOIR_INDEX(i, depth, ReSTIRSettings::numReservoirsPerPixel)];

                //Discard samples that are too different.
                float depth1 = pixelPointers[0]->depth;
                float depth2 = pixelPointers[1]->depth;
                float depthDifPct = fabs(depth1 - depth2) / ((depth1 + depth2) / 2.f);

                const float angleDif = dot(pixelPointers[0]->worldNormal, pixelPointers[1]->worldNormal);	//Between 0 and 1 (0 to 90 degrees). 
                static constexpr float MAX_ANGLE_COS = 0.72222222223f;	//Dot product is cos of the angle. If higher than this value, it's within 25 degrees.

                //Only do something if the samples are not vastly different.
                if (depthDifPct < 0.10f && angleDif > MAX_ANGLE_COS)
                {
                    //Cap sample count at 20x current to reduce temporal influence. Would grow infinitely large otherwise.
                    toCombine[0]->sampleCount = fminf(toCombine[0]->sampleCount, toCombine[1]->sampleCount * 20);

                    if (ReSTIRSettings::enableBiased)
                    {
                        CombineBiased(i, 2, toCombine, pixelPointers, a_Seed + i);
                    }
                    else
                    {
                        CombineUnbiased(i, 2, toCombine, pixelPointers, a_Seed + i);
                    }
                }
                
            }
        }
    }
}

__device__ void CombineUnbiased(int a_PixelIndex, int a_Count, Reservoir** a_Reservoirs,
                                PixelData** a_ToCombine, const std::uint32_t a_Seed)
{

    for (int depth = 0; depth < ReSTIRSettings::numReservoirsPerPixel; ++depth)
    {
        Reservoir output;
        int sampleCountSum = 0;

        for (int index = 0; index < a_Count; ++index)
        {
            auto* otherReservoir = a_Reservoirs[RESERVOIR_INDEX(index, depth, ReSTIRSettings::numReservoirsPerPixel)];
            LightSample resampled;
            Resample(&otherReservoir->sample, a_ToCombine[index], &resampled);

            const float weight = static_cast<float>(otherReservoir->sampleCount) * otherReservoir->weight * resampled.
                solidAnglePdf;

            output.Update(resampled, weight, a_Seed);

            sampleCountSum += otherReservoir->sampleCount;
        }

        output.sampleCount = sampleCountSum;

        //Weigh against other pixels to remove bias from their solid angle by re-sampling.
        int correction = 0;

        for (int index = 0; index < a_Count; ++index)
        {
            auto* otherPixel = a_ToCombine[index];
            LightSample resampled;
            Resample(&output.sample, otherPixel, &resampled);

            if (resampled.solidAnglePdf > 0)
            {
                correction += a_Reservoirs[RESERVOIR_INDEX(otherPixel->index, depth, ReSTIRSettings::numReservoirsPerPixel)]->sampleCount;
            }
        }

        //TODO Shadow ray is shot here in ReSTIR to check visibility at every resampled pixel.


        //TODO I don't understand this part fully, but it's in the pseudocode of ReSTIR. Dive into it when I have time.
        const float m = 1.f / fmaxf(static_cast<float>(correction), MINFLOAT);
        output.weight = (1.f / fmaxf(output.sample.solidAnglePdf, MINFLOAT)) * (m * output.weightSum);

        //Store the output reservoir for the pixel.
        a_Reservoirs[RESERVOIR_INDEX(a_PixelIndex, depth, ReSTIRSettings::numReservoirsPerPixel)];
    }
}

__device__ void CombineBiased(int a_PixelIndex, int a_Count, Reservoir** a_Reservoirs,
                              PixelData** a_ToCombine, const std::uint32_t a_Seed)
{
    //Loop over every depth.
    for (int depth = 0; depth < ReSTIRSettings::numReservoirsPerPixel; ++depth)
    {
        Reservoir output;
        int sampleCountSum = 0;

        //Iterate over the intersection data to combine.
        for (int i = 0; i < a_Count; ++i)
        {
            auto* pixel = a_ToCombine[i];
            auto* reservoir = a_Reservoirs[RESERVOIR_INDEX(pixel->index, depth, ReSTIRSettings::numReservoirsPerPixel)];

            LightSample resampled;
            Resample(&reservoir->sample, pixel, &resampled);

            const float weight = static_cast<float>(reservoir->sampleCount) * reservoir->weight * resampled.
                solidAnglePdf;

            assert(resampled.solidAnglePdf >= 0.f);

            output.Update(resampled, weight, a_Seed);

            sampleCountSum += reservoir->sampleCount;
        }

        //Update the sample 
        output.sampleCount = sampleCountSum;
        output.UpdateWeight();

        assert(output.weight >= 0.f && output.weightSum >= 0.f);

        //Override the reservoir for the output at this depth.
        *a_Reservoirs[RESERVOIR_INDEX(a_PixelIndex, depth, ReSTIRSettings::numReservoirsPerPixel)] = output;
    }
}

__device__ void Resample(LightSample* a_Input, const PixelData* a_PixelData, LightSample* a_Output)
{
    *a_Output = *a_Input;

    float3 pixelToLightDir = a_Input->position - a_PixelData->worldPosition;
    //Direction from pixel to light.
    const float lDistance = length(pixelToLightDir);
    //Light distance from pixel.
    pixelToLightDir /= lDistance;
    //Normalize.
    const float cosIn = clamp(dot(pixelToLightDir, a_PixelData->worldNormal), 0.f, 1.f);
    //Lambertian term clamped between 0 and 1. SurfaceN dot ToLight
    const float cosOut = clamp(dot(a_Input->normal, -pixelToLightDir), 0.f, 1.f);
    //Light normal at sample point dotted with light direction. Invert light dir for this (light to pixel instead of pixel to light)

    //Light is not facing towards the surface or too close to the surface.
    if(cosIn <= 0 || cosOut <= 0 || lDistance <= 0.01f)
    {
        a_Output->solidAnglePdf = 0;
        return;
    }

    //Geometry term G(x).
    const float solidAngle = (cosOut * a_Input->area) / (lDistance * lDistance);

    //BSDF is equal to material color for now.
    const auto brdf = MicrofacetBRDF(-pixelToLightDir, a_PixelData->directionIncoming, a_PixelData->worldNormal,
                                     a_PixelData->diffuse, a_PixelData->metallic, a_PixelData->roughness);

    //The unshadowed contribution (contributed if no obstruction is between the light and surface) takes the BRDF,
    //geometry factor and solid angle into account. Also the light radiance.
    //The only thing missing from this is the scaling with the rest of the scene based on the reservoir PDF.
    const auto unshadowedPathContribution = brdf * solidAngle * cosIn * a_Output->radiance;
    a_Output->unshadowedPathContribution = unshadowedPathContribution;

    //For the PDF, I take the unshadowed path contribution as a single float value. Average for now.
    //TODO might be better to instead take the max value? Ask Jacco.
    a_Output->solidAnglePdf = (unshadowedPathContribution.x + unshadowedPathContribution.y + unshadowedPathContribution.
        z) / 3.f;
}

__host__ void GenerateWaveFrontShadowRays(Reservoir* a_Reservoirs, PixelData* a_PixelData, MemoryBuffer* a_Atomic,
    WaveFront::ShadowRayData* a_ShadowRays)
{
    const auto numPixels = ReSTIRSettings::width * ReSTIRSettings::height;
    //Call in parallel.
    const int blockSize = CUDA_BLOCK_SIZE;
    const int numBlocks = (numPixels + blockSize - 1) / blockSize;
    GenerateWaveFrontShadowRaysInternal<<<numBlocks, blockSize>>>(a_Reservoirs, a_PixelData, a_Atomic->GetDevicePtr<int>(), a_ShadowRays);
    hipDeviceSynchronize();
}

__global__ void GenerateWaveFrontShadowRaysInternal(Reservoir* a_Reservoirs, PixelData* a_PixelData, int* a_Atomic, WaveFront::ShadowRayData* a_ShadowRays)
{
    const auto numPixels = ReSTIRSettings::width * ReSTIRSettings::height;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    //Temporary storage for the rays.
    WaveFront::ShadowRayData rays[ReSTIRSettings::numReservoirsPerPixel];

    for (int i = index; i < numPixels; i += stride)
    {
        PixelData* pixel = &a_PixelData[i];

        //Only generate shadow rays for pixels that hit a surface.
        if(pixel->depth > 0.f)
        {
            /*
             * TODO
             * Note: This currently divides the expected contribution per reservoir by the amount of reservoirs.
             * It's essentially like scaling down so that the total adds up to 100% if all shadow rays pass.
             * This does shoot one shadow ray per reservoir, but I think that's needed for accurate results.
             * If we are really desperate we could average the reservoir results and then send a single shadow ray.
             */

            for (int depth = 0; depth < ReSTIRSettings::numReservoirsPerPixel; ++depth)
            {
                //Get the contribution and scale it down based on the number of reservoirs.
                Reservoir* reservoir = &a_Reservoirs[RESERVOIR_INDEX(i, depth, ReSTIRSettings::numReservoirsPerPixel)];
                float3 contribution = (reservoir->sample.unshadowedPathContribution * (reservoir->weight / static_cast<float>(ReSTIRSettings::numReservoirsPerPixel)));

                //Generate a ray for this particular reservoir.
                int rayIndex = atomicAdd(a_Atomic, 1);
                float3 toLightDir = reservoir->sample.position - pixel->worldPosition;
                const float l = length(toLightDir);
                toLightDir /= l;

                //TODO ensure no shadow acne.
                a_ShadowRays[rayIndex] = WaveFront::ShadowRayData{pixel->worldPosition, toLightDir, l - 0.005f, contribution, WaveFront::ResultBuffer::OutputChannel::DIRECT};
            }
        }
    }
}

