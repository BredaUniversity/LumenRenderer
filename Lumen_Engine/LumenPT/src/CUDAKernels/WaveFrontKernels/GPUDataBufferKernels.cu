#include "GPUDataBufferKernels.cuh"
#include <>

CPU_ON_GPU void ResetIntersectionRayBatchMembers(
    IntersectionRayBatch* const a_RayBatch,
    unsigned int a_NumPixels,
    unsigned int a_RaysPerPixel)
{
    *const_cast<unsigned*>(&a_RayBatch->m_NumPixels) = a_NumPixels;
    *const_cast<unsigned*>(&a_RayBatch->m_RaysPerPixel) = a_RaysPerPixel;

}

CPU_ON_GPU void ResetIntersectionRayBatchData(IntersectionRayBatch* const a_RayBatch)
{

    const unsigned int bufferSize = a_RayBatch->GetSize();
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = index; i < bufferSize; i += stride)
    {
        a_RayBatch->m_Rays[i] = IntersectionRayData{};
    }

}

CPU_ON_GPU void ResetShadowRayBatchMembers(
    ShadowRayBatch* const a_ShadowRayBatch,
    unsigned int a_MaxDepth,
    unsigned int a_NumPixels,
    unsigned int a_RaysPerPixel)
{

    *const_cast<unsigned*>(&a_ShadowRayBatch->m_MaxDepth) = a_MaxDepth;
    *const_cast<unsigned*>(&a_ShadowRayBatch->m_NumPixels) = a_NumPixels;
    *const_cast<unsigned*>(&a_ShadowRayBatch->m_RaysPerPixel) = a_RaysPerPixel;

}

CPU_ON_GPU void ResetShadowRayBatchData(ShadowRayBatch* const a_ShadowRayBatch)
{

    const unsigned int bufferSize = a_ShadowRayBatch->GetSize();
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = index; i < bufferSize; i += stride)
    {

        a_ShadowRayBatch->m_ShadowRays[i] = ShadowRayData{};
    }

}

CPU_ON_GPU void ResetPixelBufferMembers(
    PixelBuffer* const a_PixelBuffer,
    unsigned a_NumPixels,
    unsigned a_ChannelsPerPixel)
{

    *const_cast<unsigned*>(&a_PixelBuffer->m_NumPixels) = a_NumPixels;
    *const_cast<unsigned*>(&a_PixelBuffer->m_ChannelsPerPixel) = a_ChannelsPerPixel;

}

CPU_ON_GPU void ResetPixelBufferData(PixelBuffer* const a_PixelBuffer)
{

    const unsigned int bufferSize = a_PixelBuffer->GetSize();
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = index; i < bufferSize; i += stride)
    {

        a_PixelBuffer->m_Pixels[i] = { 0.f, 0.f, 0.f };
    }

}