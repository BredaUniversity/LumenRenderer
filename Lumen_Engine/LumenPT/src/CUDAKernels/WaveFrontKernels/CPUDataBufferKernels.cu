#include "CPUDataBufferKernels.cuh"
#include "GPUDataBufferKernels.cuh"

CPU_ONLY void ResetIntersectionRayBatch(
    IntersectionRayBatch* const a_RayBatchDevPtr,
    unsigned int a_NumPixels,
    unsigned int a_RaysPerPixel)
{

    ResetIntersectionRayBatchMembers<<<1, 1>>>(a_RayBatchDevPtr, a_NumPixels, a_RaysPerPixel);

    const int numRays = a_NumPixels * a_RaysPerPixel;
    const int blockSize = 256;
    const int numBlocks = (numRays + blockSize - 1) / blockSize;

    ResetIntersectionRayBatchData<<<numBlocks, blockSize>>>(a_RayBatchDevPtr);

}

CPU_ONLY void ResetShadowRayBatch(
    ShadowRayBatch* a_ShadowRayBatchDevPtr,
    unsigned int a_MaxDepth,
    unsigned int a_NumPixels,
    unsigned int a_RaysPerPixel)
{

    ResetShadowRayBatchMembers<<<1, 1>>>(a_ShadowRayBatchDevPtr, a_MaxDepth, a_NumPixels, a_RaysPerPixel);

    const int numRays = a_MaxDepth * a_NumPixels * a_RaysPerPixel;
    const int blockSize = 256;
    const int numBlocks = (numRays + blockSize - 1) / blockSize;

    ResetShadowRayBatchData<<<numBlocks, blockSize>>>(a_ShadowRayBatchDevPtr);

}

CPU_ONLY void ResetPixelBuffer(
    PixelBuffer* a_PixelBufferDevPtr,
    unsigned a_NumPixels,
    unsigned a_ChannelsPerPixel)
{

    ResetPixelBufferMembers<<<1, 1>>>(a_PixelBufferDevPtr, a_NumPixels, a_ChannelsPerPixel);

    const int totalPixels = a_NumPixels * a_ChannelsPerPixel;
    const int blockSize = 256;
    const int numBlocks = (totalPixels + blockSize - 1) / blockSize;

    ResetPixelBufferData<<<numBlocks, blockSize>>>(a_PixelBufferDevPtr);

}