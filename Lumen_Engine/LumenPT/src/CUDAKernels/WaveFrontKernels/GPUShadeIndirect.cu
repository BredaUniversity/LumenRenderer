#include "hip/hip_runtime.h"
#include "GPUShadingKernels.cuh"
#include <>

#include "../../Shaders/CppCommon/RenderingUtility.h"
#include "../disney.cuh"

CPU_ON_GPU void ShadeIndirect(
    const uint3 a_ResolutionAndDepth,
    const float3 a_CameraPosition,
    const SurfaceData* a_SurfaceDataBuffer,
    const AtomicBuffer<IntersectionData>* a_Intersections,
    AtomicBuffer<IntersectionRayData>* a_IntersectionRays,
    const unsigned a_NumIntersections,
    const unsigned a_CurrentDepth,
    const unsigned a_Seed
)
{
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    //Outside of loop because multiple items can be processed by one thread. RandomFloat modifies the seed from within the loop so no repetition occurs.
    auto seed = WangHash(a_Seed + WangHash(index));

    //Loop over the amount of intersections.
    for (int i = index; i < a_NumIntersections; i += stride)
    {    	
        auto pixelIndex = a_Intersections->GetData(i)->m_PixelIndex;
        auto surfaceData = a_SurfaceDataBuffer[pixelIndex];

        //If the surface is emissive or not intersected, terminate.
        if(surfaceData.m_Emissive || surfaceData.m_IntersectionT <= 0.f)
        {
            continue;
        }

        /*
         * If the angle is too perpendicular to the normal, discard. It's too prone to floating point error which means it can't generate a reflection
         * some of the time.
         */
        if (fabs(dot(surfaceData.m_Normal, surfaceData.m_IncomingRayDirection)) < 3.f * EPSILON)
        {
            continue;
        }

        ////TODO replace with BSDF sampling (needs tangent).
        ////Calculate a diffuse reflection direction based on the surface roughness. Also retrieves the PDF for that direction being chosen on the full sphere.
        //float brdfPdf;
        //float3 bounceDirection;
        //SampleHemisphere(surfaceData.m_IncomingRayDirection, surfaceData.m_Normal, ROUGHNESS, seed, bounceDirection, brdfPdf);

        //if(brdfPdf <= 0)
        //{
        //    printf("Bad PDF: %f\n", brdfPdf);
        //}

        //assert(!isnan(bounceDirection.x) && !isnan(bounceDirection.y) && !isnan(bounceDirection.z));
        //assert(!isnan(brdfPdf));
        //assert(brdfPdf >= 0.f);

        /*
         * Terminate on inter-reflections. This is common for diffuse surfaces but not so much for others.
         * The angle of incidence makes a big difference too.
         * This could be resolved with RIS, by doing four strata on the hemisphere and taking four samples.
         * At least three of the samples will be correct.
         * This does however require multiple BRDF evaluations which is expensive.
         *
         * Because half the domain is removed, the BRDF PDF can be doubled after this passes.
         */
        //const auto bounceDotN = dot(bounceDirection, surfaceData.m_Normal);
        //if (bounceDotN <= 0.f) continue;

        ////Double BRDF PDF because half the domain is terminated above.
        //brdfPdf *= 2.f;

        /*
         * Scale the path contribution based on the PDF (over 4 PI, the entire sphere).
         * When perfectly diffuse, 1/4pi will result in exactly scaling by 4pi.
         * When mirroring, a high PDF way larger than 1 will scale down the contribution because now it comes from just one direction.
         */
        //const auto brdf = MicrofacetBRDF(invViewDir, bounceDirection, surfaceData.m_Normal, shadingData.color, METALLIC, ROUGHNESS);
        //pathContribution *= ((brdf * bounceDotN) / brdfPdf);

        float3 bounceDirection;
        float pdf = 0.f;
        bool specular = false;
        const auto bsdf = SampleBSDF(surfaceData.m_ShadingData, surfaceData.m_Normal, surfaceData.m_Normal, surfaceData.m_Tangent, -surfaceData.m_IncomingRayDirection, 1.f, RandomFloat(seed), RandomFloat(seed), RandomFloat(seed), bounceDirection, pdf, specular);
    	
        //Skip rays that have a tiny PDF.
        if (pdf <= EPSILON || isnan(pdf + bsdf.x + bsdf.y + bsdf.z))
        {
            continue;
        }

        

        //Apply russian roulette based on the BSDF. Mirrors always survive. For other surfaces take the max light transport channel and clamp at 1.
        const float russianRouletteWeight = specular ? 1.f : fminf(fmaxf(bsdf.x, fmaxf(bsdf.y, bsdf.z)), 1.f);
        const float rand = RandomFloat(seed);

        //Path termination.
        if (russianRouletteWeight < rand)
        {
            continue;
        }

        
    	
        //Scale contribution up because the path survived.
        const float russianPdf = 1.f / russianRouletteWeight;
        assert(russianPdf >= 0.f);
        assert(surfaceData.m_TransportFactor.x >= 0 && surfaceData.m_TransportFactor.y >= 0 && surfaceData.m_TransportFactor.z >= 0);
        float3 pathContribution = surfaceData.m_TransportFactor * russianPdf;
    	
        //Add the BSDF to the path contribution, along with the angle of incidence scaling.
        //Also scale by the BSDF PDF right away.
        pathContribution *= bsdf * fabsf(dot(surfaceData.m_Normal, bounceDirection)) * (1.f/pdf);
    	
        assert(pathContribution.x >= 0 && pathContribution.y >= 0 && pathContribution.z >= 0);
    	
        //Finally add the ray to the ray buffer.
        IntersectionRayData ray{pixelIndex, surfaceData.m_Position, bounceDirection, pathContribution};

        a_IntersectionRays->Add(&ray);

        continue; //Breaky
    }

}