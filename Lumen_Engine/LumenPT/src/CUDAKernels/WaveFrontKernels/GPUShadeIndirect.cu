#include "hip/hip_runtime.h"
#include "GPUShadingKernels.cuh"
#include <>

CPU_ON_GPU void ShadeIndirect(
    const uint3 a_ResolutionAndDepth,
    const SurfaceData* a_TemporalSurfaceDatBuffer,
    const SurfaceData* a_SurfaceDataBuffer,
    AtomicBuffer<IntersectionRayData>* const a_IntersectionRays,
    const TriangleLight* const a_Lights,
    const unsigned int a_NumLights,
    CDF* const a_CDF)
{

    const unsigned int numPixels = a_ResolutionAndDepth.x * a_ResolutionAndDepth.y;
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (int i = index; i < numPixels; i += stride)
    {

        //Convert the index into the screen dimensions.
        const int screenY = i / a_ResolutionAndDepth.x;
        const int screenX = i - (screenY * a_ResolutionAndDepth.x);

        //const IntersectionData& intersection = a_Intersections->GetIntersection(i, 0);
        //;       const IntersectionRayData& ray = a_PrimaryRays->GetRay(i, 0);

        //TODO russian roulette to terminate path (multiply weight with russian roulette outcome
        float russianRouletteWeight = 1.f;

        //TODO extract surface normal from intersection data.
        float3 normal = make_float3(1.f, 0.f, 0.f);

        //TODO generate random direction
        float3 dir = make_float3(1.f, 0.f, 0.f);

        //TODO get position from intersection data.
        float3 pos = make_float3(0.f, 0.f, 0.f);

        //TODO calculate BRDF to see how much light is transported.
        float3 brdf = make_float3(1.f, 1.f, 1.f); //Do this after direct shading because the material is already looked up there. Use the BRDF.
        //float3 totalLightTransport = russianRouletteWeight * ray.m_Contribution * brdf;  //Total amount of light that will end up in the camera through this path.

        //Add to the output buffer.
        //a_Output->SetRay({intersection.m_PixelIndex, pos, dir, totalLightTransport }, i, 0);
    }

}