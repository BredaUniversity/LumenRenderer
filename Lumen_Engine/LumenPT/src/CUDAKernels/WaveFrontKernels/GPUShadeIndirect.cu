#include "hip/hip_runtime.h"
#include "GPUShadingKernels.cuh"
#include <>

#include "../../Shaders/CppCommon/RenderingUtility.h"

CPU_ON_GPU void ShadeIndirect(
    const uint3 a_ResolutionAndDepth,
    const float3 a_CameraPosition,
    const SurfaceData* a_SurfaceDataBuffer,
    const AtomicBuffer<IntersectionData>* a_Intersections,
    AtomicBuffer<IntersectionRayData>* a_IntersectionRays,
    const unsigned a_NumIntersections,
    const unsigned a_CurrentDepth,
    const unsigned a_Seed
)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    //Outside of loop because multiple items can be processed by one thread. RandomFloat modifies the seed from within the loop so no repetition occurs.
    auto seed = WangHash(a_Seed + WangHash(index));

    //Loop over the amount of intersections.
    for (int i = index; i < a_NumIntersections; i += stride)
    {
        auto& intersection = *a_Intersections->GetData(i);
        auto& surfaceData = a_SurfaceDataBuffer[intersection.m_PixelIndex];

        //If the surface is emissive or not intersected, terminate.
        if(surfaceData.m_Emissive || surfaceData.m_IntersectionT <= 0.f)
        {
            continue;
        }

        //Apply russian roulette based on the surface color (dark absorbs more, so terminates sooner).
        const float russianRouletteWeight = clamp(fmaxf(surfaceData.m_Color.x, fmaxf(surfaceData.m_Color.y, surfaceData.m_Color.z)), 0.f, 1.f);
        const float rand = RandomFloat(seed);
        
        //Path termination.
        if (russianRouletteWeight < rand)
        {
            continue;
        }

        assert(surfaceData.m_TransportFactor.x >= 0 && surfaceData.m_TransportFactor.y >= 0 && surfaceData.m_TransportFactor.z >= 0);

        //Scale contribution up because the path survived.
        const float russianPdf = 1.f / russianRouletteWeight;
        float3 pathContribution = surfaceData.m_TransportFactor * russianPdf;

        /*
         * TODO: This should never happen.
         * This means the normal is pointing away from the intersection ray.
         * The surface should never have been hit.
         * Path is terminated right away.
         * TODO: Solve this elsewhere so that we don't have to have a conditional here.
         *
         * Note: Angles that are close to perpendicular need to be filtered out here too (that's what the epsilon is for) because of floating point inaccuracy
         * when sampling the hemisphere. If this is not done, PDF may be 0 for a retrieved sample.
         */
        if(dot(surfaceData.m_Normal, surfaceData.m_IncomingRayDirection) >= -2.f*EPSILON)
        {
            //TODO: this still happens.
            //printf("Warning: Surface with reverse normal hit at distance %f!\n", surfaceData.m_IntersectionT);
            continue;
        }

        //Calculate a diffuse reflection direction based on the surface roughness. Also retrieves the PDF for that direction being chosen on the full sphere.
        float brdfPdf;
        float3 bounceDirection;
        SampleHemisphere(surfaceData.m_IncomingRayDirection, surfaceData.m_Normal, surfaceData.m_Roughness, seed, bounceDirection, brdfPdf);

        if(brdfPdf <= 0)
        {
            printf("DooDoo PDF: %f\n", brdfPdf);
        }

        assert(!isnan(bounceDirection.x) && !isnan(bounceDirection.y) && !isnan(bounceDirection.z));
        assert(!isnan(brdfPdf));
        assert(brdfPdf >= 0.f);
        assert(russianPdf >= 0.f);

        /*
         * Terminate on interreflections now.
         * Note: When looping over them and simulating, a negative PDF was sometimes encountered. Very odd.
         */
        if (dot(bounceDirection, surfaceData.m_Normal) <= 0.f) continue;

        /*
         * Scale the path contribution based on the PDF (over 4 PI, the entire sphere).
         * When perfectly diffuse, 1/4pi will result in exactly scaling by 4pi.
         * When mirroring, a high PDF way larger than 1 will scale down the contribution because now it comes from just one direction.
         * TODO: Is this correct? A perfect mirror will divide by an infinitely large number. That seems counter-intuitive.
         */

        ////TODO: remove
        //const float brdfPdf = 1.f / (M_PIf * 2.f);
        //float3 bounceDirection = normalize(float3{RandomFloat(seed) * 2.f - 1.f, RandomFloat(seed) * 2.f - 1.f, RandomFloat(seed) * 2.f - 1.f});
        //if (dot(bounceDirection, surfaceData.m_Normal) < 0.f) bounceDirection *= -1.f;
        //float3 pathContribution = surfaceData.m_TransportFactor;
        //
        const auto invViewDir = -surfaceData.m_IncomingRayDirection;
        const auto brdf = MicrofacetBRDF(invViewDir, bounceDirection, surfaceData.m_Normal, surfaceData.m_Color, surfaceData.m_Metallic, surfaceData.m_Roughness);
        pathContribution *= (brdf / brdfPdf);

        //if(brdfPdf <= 0 || pathContribution.x < 0 || pathContribution.y < 0 || pathContribution.z < 0)
        //{
        //    printf("PDF is 0. This should never be picked? Seed: %u, Dot: %f, PDF: %f, Normal: %f %f %f. InvViewDir: %f %f %f\n", oldseed, dot(invViewDir, surfaceData.m_Normal), brdfPdf, surfaceData.m_Normal.x, surfaceData.m_Normal.y, surfaceData.m_Normal.z, invViewDir.x, invViewDir.y, invViewDir.z);
        //}

        assert(pathContribution.x >= 0 && pathContribution.y >= 0 && pathContribution.z >= 0);

        //Finally add the ray to the ray buffer.
        IntersectionRayData ray{intersection.m_PixelIndex, surfaceData.m_Position, bounceDirection, pathContribution};
        a_IntersectionRays->Add(&ray);
    }

}