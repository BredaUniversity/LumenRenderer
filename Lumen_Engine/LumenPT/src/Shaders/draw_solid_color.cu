#include "hip/hip_runtime.h"
//
// Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NVIDIA CORPORATION nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//


#include <cstdio>
#include <cuda/helpers.h>


#include "../../vendor/Include/sutil/vec_math.h"
#include "../../vendor/Include/Optix/optix_device.h"
#include "Optix/optix.h"
#include "CppCommon/LaunchParameters.h"
#include "CppCommon/RenderingUtility.h"
#include "CppCommon/SceneDataTableAccessor.h"

extern "C" {
__constant__ LaunchParameters params;
}

extern "C"
__global__ void __raygen__draw_solid_color()
{
    uint3 launch_index = optixGetLaunchIndex();

    RaygenData* rgd = reinterpret_cast<RaygenData*>(optixGetSbtDataPointer());

    //float3 origin = make_float3(static_cast<float>(launch_index.x) / params.m_ImageWidth, static_cast<float>(launch_index.y) / params.m_ImageHeight, 0.0f);
    //origin.x = -(origin.x * 2.0f - 1.0f); //we inverse the result, because U image coordinate points left while X vector points right
    //origin.y = -(origin.y * 2.0f - 1.0f); //we inverse the result, because V image coordinate points down while Y vector points up
    //origin = origin.x * params.U + origin.y * params.V;
    //origin += params.eye;
    //float3 dir = params.W;
	
    float3 origin = make_float3(0.f);
    float3 dir = make_float3(0.f);

    orthgraphicProjection(
        origin,
        dir,
        make_int2(launch_index.x, launch_index.y),
        make_int2(params.m_ImageWidth, params.m_ImageHeight),
        params.eye,
        params.U,
        params.V,
        params.W
    );

    perspectiveProjection(
        origin,
        dir,
        make_int2(launch_index.x, launch_index.y),
        make_int2(params.m_ImageWidth, params.m_ImageHeight),
        params.eye,
        params.U,
        params.V,
        params.W
    );

    unsigned int p0, p1, p2, p3, depth;

	//opaque trace
    optixTrace(params.m_Handle, origin, dir, 0.0f, 5000.0f, 0.0f, OptixVisibilityMask(128), OPTIX_RAY_FLAG_NONE, 0, 1, 0, p0, p1, p2, p3, depth);
	
	//volumetric trace
	optixTrace(params.m_Handle, origin, dir, 0.0f, depth, 0.0f, OptixVisibilityMask(64), OPTIX_RAY_FLAG_NONE, 0, 1, 0, p0, p1, p2, p3, depth);

    float3 col = make_float3(0.4f, 0.5f, 0.9f);

    col.x = int_as_float(p0);
    col.y = int_as_float(p1);
    col.z = int_as_float(p2);

    /*col.x = int_as_float(p3);
    col.y = int_as_float(p4);
    col.z = int_as_float(p5);*/
	
    //void* prim = params.m_SceneData->GetTableEntry(0);

    params.m_Image[launch_index.y * params.m_ImageWidth + launch_index.x] =
        make_color( col );
}

extern "C"
__global__ void __miss__MissShader()
{
    MissData* msd = reinterpret_cast<MissData*>(optixGetSbtDataPointer());

    float3 col = make_float3(0.4f, 0.5f, 0.9f);

    //optixSetPayload_0(42);
    //optixSetPayload_1(float_as_int(msd->m_Color.y));
    //optixSetPayload_2(float_as_int(msd->m_Color.z));
    //optixSetPayload_3(0);
}

extern "C"
__global__ void __closesthit__HitShader()
{
    DevicePrimitive* prim = params.m_SceneData->GetTableEntry<DevicePrimitive>(optixGetInstanceId());

    const float2 barycentrics = optixGetTriangleBarycentrics();
    float U = barycentrics.x;
    float V = barycentrics.y;
    float W = 1.0f - (U + V);
    unsigned int vertIndex = 3 * optixGetPrimitiveIndex();

    Vertex* A = &prim->m_VertexBuffer[prim->m_IndexBuffer[vertIndex + 0]];
    Vertex* B = &prim->m_VertexBuffer[prim->m_IndexBuffer[vertIndex + 1]];
    Vertex* C = &prim->m_VertexBuffer[prim->m_IndexBuffer[vertIndex + 2]];

    float2 texCoords = A->m_UVCoord * W + B->m_UVCoord * U + C->m_UVCoord * V;

    float4 smpCol = tex2D<float4>(prim->m_Material->m_DiffuseTexture, texCoords.x, texCoords.y);
    float4 finalCol = smpCol * prim->m_Material->m_DiffuseColor;

    optixSetPayload_0(float_as_int(finalCol.x));
    optixSetPayload_1(float_as_int(finalCol.y));
    optixSetPayload_2(float_as_int(finalCol.z));

    const float3 rayOrig = optixGetWorldRayOrigin();
    const float3 rayDir = optixGetWorldRayDirection();
    const float t = optixGetRayTmax();

    optixSetPayload_3(1);
	optixSetPayload_4(float_as_int(optixGetRayTmax()));
}

