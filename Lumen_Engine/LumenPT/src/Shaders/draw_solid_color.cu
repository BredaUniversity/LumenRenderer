#include "hip/hip_runtime.h"
//
// Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NVIDIA CORPORATION nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//


#include <cuda/helpers.h>

#include "Optix/optix.h"
#include "CppCommon/LaunchParameters.h"

extern "C" {
__constant__ LaunchParameters params;
}

extern "C"
__global__ void __raygen__draw_solid_color()
{
    uint3 launch_index = optixGetLaunchIndex();

    RaygenData* rgd = reinterpret_cast<RaygenData*>(optixGetSbtDataPointer());

    float3 origin = make_float3(static_cast<float>(launch_index.x) / params.m_ImageWidth, static_cast<float>(launch_index.y) / params.m_ImageHeight, 0.0f);
    origin.x = origin.x * 2.0f - 1.0f;
    origin.y = origin.y * 2.0f - 1.0f;
    float3 dir = make_float3(0.0f, 0.0f, 1.0f);

    unsigned int p0;

    optixTrace(params.m_Handle, origin, dir, 0.0f, 1000.0f, 0.0f, OptixVisibilityMask(255), OPTIX_RAY_FLAG_NONE, 0, 1, 0, p0);

    float3 col = rgd->m_Color;

    if (p0 == 1)
    {
        col = make_float3(1.0f, 1.0f, 0.0f);
    }
    else if (p0 == 3)
    {
        col = make_float3(1.0f, 0.0f, 1.0f);
    }

    params.m_Image[launch_index.y * params.m_ImageWidth + launch_index.x] =
        make_color( col );
}

extern "C"
__global__ void __miss__MissShader()
{
    MissData* msd = reinterpret_cast<MissData*>(optixGetSbtDataPointer());;
    optixSetPayload_0(msd->m_Num);
}

extern "C"
__global__ void __closesthit__HitShader()
{
    optixSetPayload_0(1);
}
