#include "hip/hip_runtime.h"
//
// Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NVIDIA CORPORATION nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//


#include <cuda/helpers.h>


#include "../../vendor/Include/sutil/vec_math.h"
#include "Optix/optix.h"
#include "CppCommon/LaunchParameters.h"

extern "C" {
__constant__ LaunchParameters params;
}

extern "C"
__global__ void __raygen__draw_solid_color()
{
    uint3 launch_index = optixGetLaunchIndex();

    RaygenData* rgd = reinterpret_cast<RaygenData*>(optixGetSbtDataPointer());

    float3 origin = make_float3(static_cast<float>(launch_index.x) / params.m_ImageWidth, static_cast<float>(launch_index.y) / params.m_ImageHeight, 0.0f);
    origin.x = -(origin.x * 2.0f - 1.0f); //we inverse the result, because U image coordinate points left while X vector points right
    origin.y = -(origin.y * 2.0f - 1.0f); //we inverse the result, because V image coordinate points down while Y vector points up
    origin = origin.x * params.U + origin.y * params.V;
    origin += params.eye;
    float3 dir = params.W;
	
    unsigned int p0, p1, p2;

    optixTrace(params.m_Handle, origin, dir, 0.0f, 1000.0f, 0.0f, OptixVisibilityMask(255), OPTIX_RAY_FLAG_NONE, 0, 1, 0, p0, p1, p2);

    float3 col = rgd->m_Color;

    if (p0 == 1)
    {
        col = make_float3(1.0f, 1.0f, 0.0f);
    }
    else if (p0 == 3)
    {
        col = make_float3(1.0f, 0.0f, 1.0f);
    }

    col = make_float3(int_as_float(p0), int_as_float(p1), int_as_float(p2));

    params.m_Image[launch_index.y * params.m_ImageWidth + launch_index.x] =
        make_color( col );
}

extern "C"
__global__ void __miss__MissShader()
{
    MissData* msd = reinterpret_cast<MissData*>(optixGetSbtDataPointer());

    optixSetPayload_0(float_as_int(msd->m_Color.z * 0.25f));
    optixSetPayload_1(float_as_int(msd->m_Color.y));
    optixSetPayload_2(float_as_int(msd->m_Color.z));
}

extern "C"
__global__ void __closesthit__HitShader()
{
    const float2 barycentrics = optixGetTriangleBarycentrics();
    HitData* msd = reinterpret_cast<HitData*>(optixGetSbtDataPointer());;
    auto col = make_float4(0.0f, 1.0f, 0.0f, 1.0f);
    auto col1 = tex2D<float4>(msd->m_TextureObject, barycentrics.x, 1 - barycentrics.y);


    optixSetPayload_0(float_as_int(col1.x));
    optixSetPayload_1(float_as_int(col1.y));
    optixSetPayload_2(float_as_int(col1.z));
}
